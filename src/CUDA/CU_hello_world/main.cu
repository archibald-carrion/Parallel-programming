
#include <hip/hip_runtime.h>
#include <stdio.h>

// host code 
void CPUFunction()
{
  printf("Hello from the CPU.\n");
}

// device code
__global__ void GPUFunction()
{
  printf("Hello from the GPU.\n");
}

int main()
{
  CPUFunction();

  GPUFunction<<<1, 1>>>();
  hipDeviceSynchronize();
}